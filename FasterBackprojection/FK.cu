#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "FK.h"

#include "CudaHelper.h"

#include "fk.cuh"
#include "fourier.cuh"
#include "transient_postprocessing.cuh"

//

void FK::reconstructDepths(NLosData* nlosData, const ReconstructionInfo& recInfo,
                           const ReconstructionBuffers& recBuffers, const TransientParameters& transientParams,
                           const std::vector<float>& depths)
{
}

void FK::reconstructVolume(
	NLosData* nlosData, 
	const ReconstructionInfo& recInfo, const ReconstructionBuffers& recBuffers, const TransientParameters& transientParams)
{
	_nlosData = nlosData;

	_perf.setAlgorithmName("fk-migration");
	_perf.tic();

	if (transientParams._compensateLaserCosDistance)
		compensateLaserCosDistance(recInfo, recBuffers);

	if (recInfo._captureSystem == CaptureSystem::Confocal)
		reconstructVolumeConfocal(nullptr, recInfo, recBuffers);
	else
		throw std::runtime_error("Unsupported capture system for LCT reconstruction.");

	_perf.toc();
	_perf.summarize();

	if (transientParams._saveMaxImage)
		FK::saveMaxImage(
			transientParams._outputFolder + transientParams._outputMaxImageName,
			recBuffers._intensity,
			glm::uvec3(nlosData->_dims[0], nlosData->_dims[1], nlosData->_dims[2]));
}

//

void FK::reconstructVolumeConfocal(float* volume, const ReconstructionInfo& recInfo, const ReconstructionBuffers& recBuffers)
{
	const glm::uvec3 volumeResolution = glm::uvec3(_nlosData->_dims[0], _nlosData->_dims[1], _nlosData->_dims[2]);
	const glm::uvec3 fftVolumeResolution = volumeResolution * 2u;

	_perf.tic("Resource allocation");

	hipfftHandle planH;
	hipfftComplex* fft = nullptr, *fftAux = nullptr;
	CudaHelper::initializeZeroBufferGPU(fft, static_cast<size_t>(fftVolumeResolution.x) * fftVolumeResolution.y * fftVolumeResolution.z);
	CudaHelper::initializeZeroBufferGPU(fftAux, static_cast<size_t>(fftVolumeResolution.x) * fftVolumeResolution.y * fftVolumeResolution.z);
	float* intensityGpu = recBuffers._intensity;

	_perf.toc();

	dim3 blockSize(16, 8, 8);
	dim3 gridSize(
		(volumeResolution.z + blockSize.x - 1) / blockSize.x,
		(volumeResolution.y + blockSize.y - 1) / blockSize.y,
		(volumeResolution.x + blockSize.z - 1) / blockSize.z
	);

	dim3 blockSizeFFT(16, 8, 8);
	dim3 gridSizeFFT(
		(fftVolumeResolution.z + blockSizeFFT.x - 1) / blockSizeFFT.x,
		(fftVolumeResolution.y + blockSizeFFT.y - 1) / blockSizeFFT.y,
		(fftVolumeResolution.x + blockSizeFFT.z - 1) / blockSizeFFT.z
	);

	// Perform forward FFT on the intensity data
	{
		_perf.tic("Pad Intensity FFT");

		padIntensityFFT_FK<<<gridSize, blockSize>>>(intensityGpu, fft, volumeResolution, fftVolumeResolution, 1.0f);

		_perf.toc();
	}

	{
		_perf.tic("FFT");

		int rank = 3;
		int n[3] = { static_cast<int>(fftVolumeResolution[0]),
					 static_cast<int>(fftVolumeResolution[1]),
					 static_cast<int>(fftVolumeResolution[2]) };

		CUFFT_CHECK(hipfftPlanMany(&planH, rank, n,
			NULL, 1, 0,
			NULL, 1, 0,
			HIPFFT_C2C, 1));
		CUFFT_CHECK(hipfftExecC2C(planH, fft, fft, HIPFFT_FORWARD));

		_perf.toc();
	}

	// Stolt interpolation
	{
		_perf.tic("Stolt");

		float width = _nlosData->_temporalWidth, range = recInfo._timeStep * static_cast<float>(recInfo._numTimeBins);
		float sqrtConst = static_cast<float>(volumeResolution.x) * range / (static_cast<float>(volumeResolution.z) * width * 4.0f);
		float maxValue = 1.0f / sqrtf(sqrtConst * sqrtConst * 2.0f + 1.0f);

		dim3 paddedBlockSizeFFT(16, 8, 8);
		dim3 paddedGridSizeFFT(
			(fftVolumeResolution.z / 2 + paddedBlockSizeFFT.x - 1) / paddedBlockSizeFFT.x,
			(fftVolumeResolution.y + paddedBlockSizeFFT.y - 1) / paddedBlockSizeFFT.y,
			(fftVolumeResolution.x + paddedBlockSizeFFT.z - 1) / paddedBlockSizeFFT.z
		);

		stoltKernel<<<paddedGridSizeFFT, paddedBlockSizeFFT>>>(fft, fftAux, volumeResolution, fftVolumeResolution, fftVolumeResolution / 2u, sqrtConst * sqrtConst, maxValue);

		_perf.toc();
	}

	// Inverse FFT
	{
		_perf.tic("Inverse FFT");

		CUFFT_CHECK(hipfftExecC2C(planH, fftAux, fftAux, HIPFFT_BACKWARD));

		_perf.toc();
	}

	// IFFT requires normalization, but it also produces very small values, so we avoid this and produce valid results by normalizing later
	//size_t fftSize = static_cast<size_t>(fftVolumeResolution.x) * fftVolumeResolution.y * fftVolumeResolution.z;
	//normalizeIFFT<<<CudaHelper::getNumBlocks(fftSize, 512), 512>>>(fftAux, fftSize, 1.0f / static_cast<float>(fftSize));

	// Inverse padding
	{
		_perf.tic("Unpad intensity FFT");

		unpadIntensityFFT_FK<<<gridSize, blockSize>>>(intensityGpu, fftAux, volumeResolution, fftVolumeResolution);

		_perf.toc();
	}

	CudaHelper::free(fft);
	CudaHelper::free(fftAux);
	CUFFT_CHECK(hipfftDestroy(planH));
}

void FK::reconstructVolumeExhaustive(float* volume, const ReconstructionInfo& recInfo)
{
}

