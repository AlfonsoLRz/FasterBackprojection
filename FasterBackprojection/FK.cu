#include "stdafx.h"
#include "FK.h"

#include "CudaHelper.h"
#include "fk.cuh"
#include "fourier.cuh"
#include "transient_postprocessing.cuh"

//

void FK::reconstructDepths(NLosData* nlosData, const ReconstructionInfo& recInfo,
                           const ReconstructionBuffers& recBuffers, const TransientParameters& transientParams,
                           const std::vector<float>& depths)
{
}

void FK::reconstructVolume(
	NLosData* nlosData, 
	const ReconstructionInfo& recInfo, const ReconstructionBuffers& recBuffers, const TransientParameters& transientParams)
{
	_nlosData = nlosData;

	_perf.setAlgorithmName("fk-migration");
	_perf.tic();

	compensateLaserCosDistance(transientParams, recInfo, recBuffers);

	if (recInfo._captureSystem == CaptureSystem::Confocal)
		reconstructVolumeConfocal(nullptr, recInfo, recBuffers);
	else
		throw std::runtime_error("Unsupported capture system for LCT reconstruction.");

	const glm::uvec3 volumeResolution = glm::uvec3(nlosData->_dims[0], nlosData->_dims[1], nlosData->_dims[2]);
	float* volumeGpu = recBuffers._intensity;

	// Post-process the activation matrix
	_perf.tic("Post-processing");
	_postprocessingFilters[transientParams._postprocessingFilterType]->compute(volumeGpu, volumeResolution, transientParams);
	_perf.toc();

	normalizeMatrix(volumeGpu, volumeResolution.x * volumeResolution.y * volumeResolution.z);

	_perf.toc();
	_perf.summarize();

	// Save volume & free resources
	if (transientParams._saveReconstructedBoundingBox)
		saveReconstructedAABB(
			transientParams._outputFolder + transientParams._outputAABBName, volumeGpu, 
			volumeResolution.x * volumeResolution.y * volumeResolution.z);

	if (transientParams._saveMaxImage)
		FK::saveMaxImage(
			transientParams._outputFolder + transientParams._outputMaxImageName,
			volumeGpu,
			glm::uvec3(nlosData->_dims[0], nlosData->_dims[1], nlosData->_dims[2]));
}

//

void FK::reconstructVolumeConfocal(float* volume, const ReconstructionInfo& recInfo, const ReconstructionBuffers& recBuffers)
{
	const glm::uvec3 volumeResolution = glm::uvec3(_nlosData->_dims[0], _nlosData->_dims[1], _nlosData->_dims[2]);
	const glm::uvec3 fftVolumeResolution = glm::vec3(volumeResolution) * glm::vec3(2);

	_perf.tic("Resource allocation");

	hipStream_t stream1, stream2;
	CudaHelper::createStreams({ &stream1, &stream2 });

	hipfftHandle planH;
	hipfftComplex* fft = nullptr, *fftAux = nullptr;
	CudaHelper::initializeZeroBufferAsync(
		fft, static_cast<size_t>(fftVolumeResolution.x) * fftVolumeResolution.y * fftVolumeResolution.z,
		stream1
	);
	CudaHelper::initializeBufferAsync(
		fftAux, static_cast<size_t>(fftVolumeResolution.x) * fftVolumeResolution.y * fftVolumeResolution.z,
		static_cast<hipfftComplex*>(nullptr), stream2
	);
	float* intensityGpu = recBuffers._intensity;

	_perf.toc();

	dim3 blockSize(16, 8, 8);
	dim3 gridSize(
		(volumeResolution.z + blockSize.x - 1) / blockSize.x,
		(volumeResolution.y + blockSize.y - 1) / blockSize.y,
		(volumeResolution.x + blockSize.z - 1) / blockSize.z
	);

	dim3 blockSizeFFT(16, 8, 8);
	dim3 gridSizeFFT(
		(fftVolumeResolution.z + blockSizeFFT.x - 1) / blockSizeFFT.x,
		(fftVolumeResolution.y + blockSizeFFT.y - 1) / blockSizeFFT.y,
		(fftVolumeResolution.x + blockSizeFFT.z - 1) / blockSizeFFT.z
	);

	// Perform forward FFT on the intensity data
	CudaHelper::waitFor({ &stream1 });
	{
		_perf.tic("Pad Intensity FFT");

		float divisor = 1.0f / static_cast<float>(volumeResolution.z);
		fk::padIntensityFFT_FK<<<gridSize, blockSize>>>(intensityGpu, fft, volumeResolution, fftVolumeResolution, divisor);

		_perf.toc();
	}

	{
		_perf.tic("FFT");

		int rank = 3;
		int n[3] = { static_cast<int>(fftVolumeResolution[0]),
					 static_cast<int>(fftVolumeResolution[1]),
					 static_cast<int>(fftVolumeResolution[2]) };

		CUFFT_CHECK(hipfftPlanMany(&planH, rank, n,
			NULL, 1, 0,
			NULL, 1, 0,
			HIPFFT_C2C, 1));
		CUFFT_CHECK(hipfftExecC2C(planH, fft, fft, HIPFFT_FORWARD));

		_perf.toc();
	}

	// Stolt interpolation
	CudaHelper::waitFor({ &stream2 });
	{
		_perf.tic("Stolt");

		float width = _nlosData->_wallWidth, range = recInfo._timeStep * static_cast<float>(recInfo._numTimeBins);
		float sqrtConst = static_cast<float>(volumeResolution.x) * range / (static_cast<float>(volumeResolution.z) * width * 4.0f);

		dim3 paddedBlockSizeFFT(8, 8, 8);
		dim3 paddedGridSizeFFT(
			(fftVolumeResolution.z / 2 + paddedBlockSizeFFT.x - 1) / paddedBlockSizeFFT.x,
			(fftVolumeResolution.y + paddedBlockSizeFFT.y - 1) / paddedBlockSizeFFT.y,
			(fftVolumeResolution.x + paddedBlockSizeFFT.z - 1) / paddedBlockSizeFFT.z
		);

		fk::stoltKernel<<<paddedGridSizeFFT, paddedBlockSizeFFT>>>(
			fft, fftAux,
			fftVolumeResolution, fftVolumeResolution / 2u, 
			sqrtConst * sqrtConst);

		_perf.toc();
	}

	// Inverse FFT
	{
		_perf.tic("Inverse FFT");

		CUFFT_CHECK(hipfftExecC2C(planH, fftAux, fftAux, HIPFFT_BACKWARD));

		_perf.toc();
	}

	// IFFT requires normalization, but it also produces very small values, so we avoid this and produce valid results by normalizing later
	//size_t fftSize = static_cast<size_t>(fftVolumeResolution.x) * fftVolumeResolution.y * fftVolumeResolution.z;
	//normalizeIFFT<<<CudaHelper::getNumBlocks(fftSize, 512), 512>>>(fftAux, fftSize, 1.0f / static_cast<float>(fftSize));

	// Inverse padding
	{
		_perf.tic("Unpad intensity FFT");

		fk::unpadIntensityFFT_FK<<<gridSize, blockSize>>>(intensityGpu, fftAux, volumeResolution, fftVolumeResolution);

		_perf.toc();
	}

	spdlog::info("Allocated memory: {} MB", CudaHelper::getAllocatedMemory() / static_cast<size_t>(1024 * 1024));

	CudaHelper::freeAsync(fft, stream1);
	CudaHelper::freeAsync(fftAux, stream2);
	CUFFT_CHECK(hipfftDestroy(planH));
	CudaHelper::waitFor({ &stream1, &stream2 });
	CudaHelper::destroyStreams({ &stream1, &stream2 });
}

void FK::reconstructVolumeExhaustive(float* volume, const ReconstructionInfo& recInfo)
{
}

